#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudafilters.hpp>
#include <stdio.h>
#include "iostream"
#include "hip/hip_runtime.h"
#include "admm_tools.cuh"
using namespace std;
using namespace cv;

int main(){
    const Size KERNEL_SIZE(15, 15);
    const double SIGMA = 1.5;
    const int SCALE = 100;
    const double LAMBDA = sqrt(2) - 1;

    // image resize, Gaussian blur and noise adding of original image

    Mat img = imread("lena.bmp");
    Mat img_gray_tmp;
    Mat_<double> img_gray;
    cvtColor(img, img_gray_tmp, CV_RGB2GRAY);
    img_gray_tmp.convertTo(img_gray, CV_64FC1, 1.0/255.0);
    img.release();
    img_gray_tmp.release();
    resize(img_gray, img_gray, Size(), 1.0 / 4, 1.0 / 4, INTER_LINEAR);
    //namedWindow("img1", WINDOW_NORMAL);
    //imshow("img1", img_gray);
    //waitKey(3000);
    GaussianBlur(img_gray, img_gray, KERNEL_SIZE, SIGMA, BORDER_WRAP);
    //namedWindow("img2", WINDOW_NORMAL);
    //imshow("img2", img_gray);
    //waitKey(3000);
    double* value_max_img = new double[1];
    cuda::minMax(img_gray, NULL, value_max_img);
    Mat_<double> img_noise(img_gray.rows, img_gray.cols, CV_64FC1);
    randn(img_noise, 0, value_max_img[0]);
    cuda::multiply(img_noise, value_max_img[0] / SCALE, img_noise);
    cuda::add(img_gray, img_noise, img_gray);
    //namedWindow("img3", WINDOW_NORMAL);
    //imshow("img3", img_gray);
    //waitKey(3000);
    img_noise.release();
    free(value_max_img);

    // generate matrix A and W

    Mat_<double> A = Mat::zeros(img_gray.rows * img_gray.cols, img_gray.rows * img_gray.cols, CV_64FC1);
    if (img_gray.isContinuous() == 0){
        throw "memory of gray image is not continuous, cannot use cuda!";
    }
    Mat_<double> W = Mat::zeros(img_gray.rows * img_gray.cols, img_gray.rows * img_gray.cols, CV_64FC1);
    if (img_gray.isContinuous() == 0){
        throw "memory of gray image is not continuous, cannot use cuda!";
    }
    double* A_host = A.ptr<double>(0);
    double* A_device;
    hipMalloc((void**)&A_device, sizeof(double) * A.cols * A.rows);
    hipMemcpy(A_device, A_host, sizeof(double) * A.cols * A.rows, hipMemcpyHostToDevice);
    double* W_host = W.ptr<double>(0);
    double* W_device;
    hipMalloc((void**)&W_device, sizeof(double) * W.cols * W.rows);
    hipMemcpy(W_device, W_host, sizeof(double) * W.cols * W.rows, hipMemcpyHostToDevice);
    Mat_<double> Gauss = Mat::zeros(15, 15, CV_64FC1);
    Gauss.at<double>(7, 7) = 1;
    GaussianBlur(Gauss, Gauss, KERNEL_SIZE, SIGMA, BORDER_WRAP);
    double* Gauss_device;
    hipMalloc((void**)&Gauss_device, sizeof(double) * Gauss.cols * Gauss.rows);
    hipMemcpy(Gauss_device, Gauss.ptr<double>(0), sizeof(double) * Gauss.cols * Gauss.rows, hipMemcpyHostToDevice);
    dim3 thread_perblock(Gauss.rows, Gauss.cols);
    compute_A<<<A.rows, thread_perblock>>>(A_device, Gauss_device, A.rows, A.cols, img_gray.rows, img_gray.cols, Gauss.rows, Gauss.cols);
    compute_W<<<W.rows, 1>>>(W_device, W.rows, W.cols, img_gray.rows, img_gray.cols, LAMBDA);
    hipMemcpy(A_host, A_device, sizeof(double) * A.cols * A.rows, hipMemcpyDeviceToHost);
    hipMemcpy(W_host, W_device, sizeof(double) * W.cols * W.rows, hipMemcpyDeviceToHost);
    hipFree(Gauss_device);
    hipFree(A_device);
    hipFree(W_device);
    Gauss.release();
}
