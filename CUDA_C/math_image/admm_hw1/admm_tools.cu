#include "admm_tools.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

void compute_A(double* A_device, double* Gauss_device, int rows, int cols, int ori_rows, int ori_cols, int Gauss_rows, int Gauss_cols)
{
    int i = blockIdx.y;
    int j = blockIdx.x;
    int k1 = threadIdx.y;
    int k2 = threadIdx.x;
    int len_x = (Gauss_rows - 1) / 2;
    int len_y = (Gauss_cols - 1) / 2;
    if (i < rows && j < cols)
    {
        int a = j / ori_cols;
        int b = j % ori_cols;
        int a1 = a - len_x;
        int a2 = a + len_x;
        int b1 = b - len_y;
        int b2 = b + len_y;
        int k_a = k1 + a1;
        int k_b = k2 + b1;
        if (k_a >= 0 && k_a < ori_rows && k_b >= 0 && k_b < ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[(k_a * ori_cols + k_b) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// inner situation
        if (k_a < 0 && k_b >= 0 && k_b < ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((ori_rows + k_a) * ori_cols + k_b) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// top edge
        if (k_a >= ori_rows && k_b >= 0 && k_b < ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a - ori_rows) * ori_cols + k_b) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// bottom edge
        if (k_a >= 0 && k_a < ori_rows && k_b < 0 && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[(k_a * ori_cols + (k_b + ori_cols)) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// laft edge
        if (k_a >= 0 && k_a < ori_rows && k_b >= ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[(k_a * ori_cols + (k_b - ori_cols)) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// right edge
        if (k_a < 0 && k_b < 0 && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a + ori_rows) * ori_cols + (k_b + ori_cols)) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// left top corner
        if (k_a < 0 && k_b >= ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a + ori_rows) * ori_cols + (k_b - ori_cols)) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// right top corner
        if (k_a >= ori_rows && k_b < 0 && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a - ori_rows) * ori_cols + (k_b + ori_cols)) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// left bottom corner
        if (k_a >= ori_rows && k_b >= ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a - ori_rows) * ori_cols + (k_b - ori_cols)) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// right bottom corner
    }
}


void compute_W(double* W_device, int rows, int cols, int ori_rows, int ori_cols, double LAMBDA)
{
    int i = blockIdx.y;
    int j = blockIdx.x;
    int a = i / ori_cols;
    int b = i % ori_cols;
    if (a > 0 && a < ori_rows - 1 && b > 0 && b < ori_cols - 1)
    {
        W_device[((a - 1) * ori_cols + b) * cols + j] = LAMBDA / 2.0;
        W_device[((a + 1) * ori_cols + b) * cols + j] = LAMBDA / 2.0;
        W_device[(a * ori_cols + (b - 1)) * cols + j] = LAMBDA / 2.0;
        W_device[(a * ori_cols + (b + 1)) * cols + j] = LAMBDA / 2.0;
        W_device[((a - 1) * ori_cols + (b - 1)) * cols + j] = (1 - LAMBDA) / 2.0;
        W_device[((a - 1) * ori_cols + (b + 1)) * cols + j] = (1 - LAMBDA) / 2.0;
        W_device[((a + 1) * ori_cols + (b - 1)) * cols + j] = (1 - LAMBDA) / 2.0;
        W_device[((a + 1) * ori_cols + (b + 1)) * cols + j] = (1 - LAMBDA) / 2.0;
    }
}
