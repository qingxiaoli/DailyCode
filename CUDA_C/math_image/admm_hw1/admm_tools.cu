#include "hip/hip_runtime.h"
#include "admm_tools.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void compute_A(double* A_device, double* Gauss_device, int rows, int cols, int ori_rows, int ori_cols, int Gauss_rows, int Gauss_cols)
{
    int j = blockIdx.x;
    int k1 = threadIdx.y;
    int k2 = threadIdx.x;
    int len_x = (Gauss_rows - 1) / 2;
    int len_y = (Gauss_cols - 1) / 2;
    if (j < cols)
    {
        int b = j / ori_rows;
        int a = j % ori_rows;
        int a1 = a - len_x;
        int b1 = b - len_y;
        int k_a = k1 + a1;
        int k_b = k2 + b1;
        if (k_a >= 0 && k_a < ori_rows && k_b >= 0 && k_b < ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[(k_b * ori_rows + k_a) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// inner situation
        if (k_a < 0 && k_b >= 0 && k_b < ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((ori_rows + k_a) + k_b * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// top edge
        if (k_a >= ori_rows && k_b >= 0 && k_b < ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a - ori_rows) + k_b * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// bottom edge
        if (k_a >= 0 && k_a < ori_rows && k_b < 0 && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[(k_a + (k_b + ori_cols) * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// laft edge
        if (k_a >= 0 && k_a < ori_rows && k_b >= ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[(k_a + (k_b - ori_cols) * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// right edge
        if (k_a < 0 && k_b < 0 && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a + ori_rows) + (k_b + ori_cols) * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// left top corner
        if (k_a < 0 && k_b >= ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a + ori_rows) + (k_b - ori_cols) * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// right top corner
        if (k_a >= ori_rows && k_b < 0 && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a - ori_rows) + (k_b + ori_cols) * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// left bottom corner
        if (k_a >= ori_rows && k_b >= ori_cols && k1 < Gauss_rows && k2 < Gauss_cols)
        {
          A_device[((k_a - ori_rows) + (k_b - ori_cols) * ori_rows) * cols + j] = Gauss_device[k1 * Gauss_cols + k2];
        }// right bottom corner
        __syncthreads();
    }
}


__global__ void compute_W(double* W_device, int rows, int cols, int ori_rows, int ori_cols, double LAMBDA)
{
    int j = blockIdx.x;
    int a = j / ori_rows;
    int b = j % ori_rows;
    if (a > 0 && a < ori_cols - 1 && b > 0 && b < ori_rows - 1)
    {
        W_device[(j - 1) * cols + j] = LAMBDA / 2.0;
        W_device[(j + 1) * cols + j] = LAMBDA / 2.0;
        W_device[(j - ori_rows) * cols + j] = LAMBDA / 2.0;
        W_device[(j + ori_rows) * cols + j] = LAMBDA / 2.0;
        W_device[(j - ori_rows - 1) * cols + j] = (1 - LAMBDA) * sqrt(2.0) / 4.0;
        W_device[(j - ori_rows + 1) * cols + j] = (1 - LAMBDA) * sqrt(2.0) / 4.0;
        W_device[(j + ori_rows - 1) * cols + j] = (1 - LAMBDA) * sqrt(2.0) / 4.0;
        W_device[(j + ori_rows + 1) * cols + j] = (1 - LAMBDA) * sqrt(2.0) / 4.0;
    }
}
